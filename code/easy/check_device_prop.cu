//check device properties
#include <iostream>
using namespace std;

int main(void){
	hipDeviceProp_t prop;
	
	int count;
	hipGetDeviceCount(&count);
	for(int i=0; i<count; i++){
		hipGetDeviceProperties(&prop, i);
		cout << " --- General Information for device " << i << " --- " << endl;
		cout << "Name: " << prop.name << endl;
		cout << "Compute capability: " << prop.major << "." << prop.minor << endl;
		cout << "Clock rate: " << prop.clockRate << endl;
		cout << "Device copy overlap: ";
		if (prop.deviceOverlap)
			cout << "Enabled" << endl;
		else
			cout << "Disabled" << endl;
		cout << "Kernel execition timeout: ";
		if (prop.kernelExecTimeoutEnabled)
			cout << "Enabled" << endl;
		else
			cout << "Disabled" << endl;
			
		cout << " --- Memory Information for device " << i << "---" << endl;
		cout << "Total global mem: " << prop.totalGlobalMem << endl;
		cout << "TOtal constant Mem: " << prop.totalConstMem << endl;
		cout << "Max mem pitch: " << prop.memPitch << endl;
		cout << "Texture Alignment: " << prop.textureAlignment << endl;
		cout << " --- MP Information for device " << i << " --- " << endl;
		cout << "Multiprocessor count: " << prop.multiProcessorCount << endl;
		cout << "Shared mem per mp: " << prop.sharedMemPerBlock << endl;
		cout << "Registers per mp: " << prop.regsPerBlock << endl;
		cout << "Threads in warp: " << prop.warpSize << endl;
		cout << "Max threads per block: " << prop.maxThreadsPerBlock << endl;
		cout << "Max thread dimensions: " << prop.maxThreadsDim[0] << " "\
										  << prop.maxThreadsDim[1] << " "\
										  << prop.maxThreadsDim[2] << " " << endl;
		cout << "Max grid dimensions: "   << prop.maxGridSize[0] << " "\
										  << prop.maxGridSize[1] << " "\
										  << prop.maxGridSize[2] << " " << endl;
		cout << endl;								  
	}
}
