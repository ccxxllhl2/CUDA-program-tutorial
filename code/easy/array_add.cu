
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#define N 10

__global__ void add(int *a, int *b, int *c){
	int tid = blockIdx.x;
	if(tid < N)
		c[tid] = a[tid] + b[tid];
}

int main(void){
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	
	//allocate memory on GPU
	hipMalloc((void**)&dev_a, N*sizeof(int));
	hipMalloc((void**)&dev_b, N*sizeof(int));
	hipMalloc((void**)&dev_c, N*sizeof(int));
	
	//set value for vars
	for(int i=0; i<N; i++){
		a[i] = -i;
		b[i] = i*i;
	}
	
	//copy arrays to GPU
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	//N threads on GPU
	add<<<N,1>>>(dev_a, dev_b, dev_c);
	
	//copy array c from GPU to CPU
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	
	//show result
	for(int i=0; i<N; i++){
		cout << a[i] << "+" << b[i] << "=" << c[i] << endl;
	}
	
	//free memory on GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	return 0;
}
